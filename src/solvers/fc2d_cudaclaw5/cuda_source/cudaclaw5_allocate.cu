#include "cudaclaw5_allocate.h"

#include <fc2d_cudaclaw5_options.h>

#include <fclaw_global.h>
#include <fclaw_patch.h>
#include <fclaw_clawpatch.h>
#include <fclaw_clawpatch_options.h>
#include <fclaw_timer.h>



void cudaclaw5_allocate_fluxes(struct fclaw_global *glob,
                               struct fclaw_patch *patch)
{
    const fclaw_clawpatch_options_t *claw_opt = fclaw_clawpatch_get_options(glob);
    int mx = claw_opt->mx;
    int my = claw_opt->my;
    int mbc = claw_opt->mbc;
    int meqn = claw_opt->meqn;
    int maux = claw_opt->maux;

    fc2d_cudaclaw5_options_t* cuda_opt = fc2d_cudaclaw5_get_options(glob);
    int mwaves = cuda_opt->mwaves;

    cudaclaw5_fluxes_t *fluxes = FCLAW_ALLOC(cudaclaw5_fluxes,1);

    size_t size = (2*mbc+mx)*(2*mbc+my)*sizeof(double);
    fluxes->num_bytes        = meqn*size;
    fluxes->num_bytes_aux    = maux*size;
    fluxes->num_bytes_waves  = mwaves*meqn*size;
    fluxes->num_bytes_speeds = mwaves*size;

    /* Assumption here is that hipMalloc is a synchronous call */
    fclaw_timer_start (&glob->timers[FCLAW_TIMER_CUDA_ALLOCATE]); 
          
    hipMalloc((void**)&fluxes->qold_dev,   fluxes->num_bytes);
    hipMalloc((void**)&fluxes->fm_dev,     fluxes->num_bytes);
    hipMalloc((void**)&fluxes->fp_dev,     fluxes->num_bytes);
    hipMalloc((void**)&fluxes->gm_dev,     fluxes->num_bytes);
    hipMalloc((void**)&fluxes->gp_dev,     fluxes->num_bytes);
    hipMalloc((void**)&fluxes->aux_dev,    fluxes->num_bytes_aux);
    hipMalloc((void**)&fluxes->waves_dev,  fluxes->num_bytes_waves);
    hipMalloc((void**)&fluxes->speeds_dev, fluxes->num_bytes_speeds);

    fclaw_timer_stop (&glob->timers[FCLAW_TIMER_CUDA_ALLOCATE]);    

    fclaw_patch_set_user_data(glob,patch,fluxes);
}

void cudaclaw5_deallocate_fluxes(fclaw_global_t *glob,
                                 fclaw_patch_t *patch)
{
    cudaclaw5_fluxes_t *fluxes = (cudaclaw5_fluxes_t*) 
               fclaw_patch_get_user_data(glob,patch);

    FCLAW_ASSERT(fluxes != NULL);

    /* Assumption here is that hipFree is a synchronous call */
    fclaw_timer_start (&glob->timers[FCLAW_TIMER_CUDA_ALLOCATE]);       
    hipFree(fluxes->qold_dev);
    hipFree(fluxes->fm_dev);
    hipFree(fluxes->fp_dev);
    hipFree(fluxes->gm_dev);
    hipFree(fluxes->gp_dev);
    hipFree(fluxes->aux_dev);
    hipFree(fluxes->waves_dev);
    hipFree(fluxes->speeds_dev);
    fclaw_timer_stop (&glob->timers[FCLAW_TIMER_CUDA_ALLOCATE]);    

    FCLAW_FREE((void*) fluxes);
}

